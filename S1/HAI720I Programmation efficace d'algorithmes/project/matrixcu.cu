
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

#include <time.h>

#define cudaCheckError() {                                                                       \
        hipError_t e=hipGetLastError();                                                        \
        if(e!=hipSuccess) {                                                                     \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));        \
            exit(EXIT_FAILURE);                                                                  \
        }                                                                                        \
    }

__global__ void kernel(float *a,float *b, float *c, int N) {
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  if(i<N)
    c[i]=a[i]+b[i];
}

__global__ void addMatrices(float *a,float *b, float *c, int N, int M) {
  int col=blockIdx.x*blockDim.x+threadIdx.x;
  int row=blockIdx.y*blockDim.y+threadIdx.y;
  int idx = row*N + col;

  if( (row<N) && (col <N))
    c[idx]=a[idx]+b[idx];

}

void caddMatrices(float *a, float *b, float*c,int N, int M)
{

  for(int i =0;i<N*M;i++)
  {

    for(int j=0;j<M;j++)
    { 
      int idx = N*i + j;
        c[idx]=a[idx]+b[idx];
    }

  }
}

int main(int argc, char *argv[]) {
  srand( time( NULL ) );
  
  if(argc < 2) { 
    return 0;
 }
  int n  = atoi(argv[1]);
  int m  = atoi(argv[2]);

  int threads=4;
  int size = m*n * sizeof(float);

  float *d_A, *d_B, *d_C;
  float *h_A, *h_B, *h_C;
  float * result;

  dim3 DimGrid((n-1)/threads+1, (m-1)/threads+1,1);
  dim3 DimBlock(threads, threads,1);

  hipMalloc((void **) &d_A, size);    
  hipMalloc((void **) &d_B, size);
  hipMalloc((void **) &d_C, size);

  h_A = (float*)malloc(size);
  h_B = (float*)malloc(size);
  h_C = (float*)malloc(size);
  result = (float*)malloc(size);


  for(int i=0;i<n*m;i++)
  {
    h_A[i]=(float)(rand() % 65535);
    h_B[i]=(float)(rand() % 65535);
  }
  /*for(int i=0;i<n*m;i++)
    printf("a: %f : b: %f \n",h_A[i],h_B[i]);*/

  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);


  addMatrices<<<DimGrid,DimBlock>>>(d_A,d_B,d_C, n,m);
  hipDeviceSynchronize();


  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
  caddMatrices(h_A,h_B,result,n,m);

  /*for(int i=0;i<(n*m);i++)
    printf("%f\n",h_C[i]);

  for(int i=0;i<(n*m);i++)
    printf("%f\n",result[i]);*/
  int b=1;
  for(int i=0;i<n*m;i++)
  {
    if(h_C[i]!=result[i])
      b =0;
  }
  
  if(b)
    printf("Tout ok !");


  hipFree(d_A); hipFree(d_B); hipFree (d_C);
  free(h_A);  free(h_B);  free(h_C); free(result);



  cudaCheckError();
  return 0;
}
