#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <time.h>

#define cudaCheckError() {                                                                       \
        hipError_t e=hipGetLastError();                                                        \
        if(e!=hipSuccess) {                                                                     \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));        \
            exit(EXIT_FAILURE);                                                                  \
        }                                                                                        \
    }

__global__ void kernel(float *a,float *b, float *c, int N) {
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  if(i<N)
    c[i]=a[i]+b[i];
}

__global__ void addMatrices(float *a,float *b, float *c, int N) {
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  if(i<N)
    c[i]=a[i]+b[i];
}

float * addMatrices(float *a, float *b ,int N, int M)
{

  float c* =(float*)malloc(N*M*sizeof(float));
  for(int i =0;i<N*M;i++)
  {
    int j = N*i+i;
    c[j]=a[j]+b[j];

  }
  return c;
}
/*

Reponses aux questions :
1/ 
1024 -> 1*1024 OU 2*512 OU 4*256 OU 8*128 OU 16*64 OU 32*32
1023 -> comme 1024
1025 -> 2*1024 OU OU 3*512 OU 4*256 OU 9*128 OU 17*64 OU 33*32
2/ 
taille grid = (n+127)/128

3/ 1+128*k
*/
int main(int argc, char *argv[]) {
  srand( time( NULL ) );
  
  if(argc < 1) { 
    return 0;
 }
  int n  = atoi(argv[1]);
  //int n=4097;
  int threads=1024;
  int blocks=(n+threads-1)/threads;
  int size = n * sizeof(float);

  float *d_A, *d_B, *d_C;
  float *h_A, *h_B, *h_C;

  h_A = (float*)malloc(n*sizeof(float));
  h_B = (float*)malloc(n*sizeof(float));
  h_C = (float*)malloc(n*sizeof(float));


  for(int i=0;i<n;i++)
  {
    h_A[i]=(float)(rand() % 65535);
    h_B[i]=(float)(rand() % 65535);
  }

  hipMalloc((void **) &d_A, size);    
  hipMalloc((void **) &d_B, size);
  hipMalloc((void **) &d_C, size);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
/*
  hipMallocManaged(&a,N*sizeof(int));
  hipMallocManaged(&b,N*sizeof(int));
  hipMallocManaged(&c,N*sizeof(int));
*/



  kernel<<<blocks,threads>>>(d_A,d_B,d_C, size);
  hipDeviceSynchronize();


  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
  /*for(int i=0;i<n;i++)
    printf("%f\n",h_C[i]);*/
  hipFree(d_A); hipFree(d_B); hipFree (d_C);
  cudaCheckError();
  return 0;
}
