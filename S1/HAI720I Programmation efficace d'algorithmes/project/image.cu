#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include "image-pnm.h"
#include <time.h>

#define cudaCheckError() {                                                                       \
        hipError_t e=hipGetLastError();                                                        \
        if(e!=hipSuccess) {                                                                     \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));        \
            exit(EXIT_FAILURE);                                                                  \
        }                                                                                        \
    }

__global__ blur_pbm_kernel(uint8 t *ImgOut, const uint8 t* imgIn, int width, int height, int b)
{
  int col=blockIdx.x*blockDim.x+threadIdx.x;
  int row=blockIdx.y*blockDim.y+threadIdx.y;

  int idx = row*width + col;

  if( (row<width) && (col <height))
    c[idx]=a[idx]+b[idx];
  {
    if(row>=b && col>=b)
    {
       int s=0;
      for(int k = i-b; k<i+b ; k++)
        for(int l = j-b ; l<j+b ; l++)
          s+= imgIn(k,l);
      s= s/((2*b+1)*(2*b+1));
      ImgOut[i*dst.width()+j]=s;
    }

  }
  
}

void blur_pgm(Image<int> src, Image<int> dst, int b)
{
  int *data;
  data = dst.getData();
  for(int i =b; i < dst.height();i++)
    for(int j = b ; j< dst.width();j++)
    {
      int s=0;

      for(int k = i-b; k<i+b ; k++)
        for(int l = j-b ; l<j+b ; l++)
          s+= src.get(k,l);
      s= s/((2*b+1)*(2*b+1));
      data[i*dst.width()+j]=s;
    }
    dst.write("image-grey2.pgm");

}

int main(int argc, char *argv[]) {
  srand( time( NULL ) );
  
  if(argc < 2) { 
    return 0;
 }
  int n  = atoi(argv[1]);
  int m  = atoi(argv[2]);
  
  uint8 *d_A, *d_B;
  uint8 *h_A, *h_B;

  //int * data;
  Image<int> src = Image<int>("P2", 512,512 );
  Image<int> dst = Image<int>("P2", 512,512 );
  n  = 512;
  m  = 512;

  int threads=4;
  int size = m*n * sizeof(uint8);

  dim3 DimGrid((n-1)/threads+1, (m-1)/threads+1,1);
  dim3 DimBlock(threads, threads,1);

  hipMalloc((void **) &d_A, size);    
  hipMalloc((void **) &d_B, size);

  h_A = (uint8*)malloc(size);
  h_B = (uint8*)malloc(size);

  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);


  blur_pbm_kernel<<<DimGrid,DimBlock>>>(d_A,d_B, n,m,25);
  hipDeviceSynchronize();

  hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);


  src.read("image-grey.pgm");
  //data = src.getData(); 
  /*for(int i =0 ; i < 512*512;i++)
    //printf("%d",data[i]);
    data[i]*=2;*/
  blur_pgm(src,  dst, 25);
  
  printf("ok");
    hipFree(d_A); hipFree(d_B);
  free(h_A);  free(h_B); 


  cudaCheckError();
  return 0;
}
