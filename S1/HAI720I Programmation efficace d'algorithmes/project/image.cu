#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include "image-pnm.h"
#include <time.h>
#include <iostream>
#include <fstream>

#define cudaCheckError() {                                                                       \
        hipError_t e=hipGetLastError();                                                        \
        if(e!=hipSuccess) {                                                                     \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));        \
            exit(EXIT_FAILURE);                                                                  \
        }                                                                                        \
    }

__global__ void blur_pbm_kernel(uint8_t *ImgOut, const uint8_t* imgIn, int width, int height, int b)
{
  
  int col=blockIdx.x*blockDim.x+threadIdx.x;
  int row=blockIdx.y*blockDim.y+threadIdx.y;

  int idx = row*width + col;

  int s;
  if( (row<width) && (col <height))
  {
    s =0;

    for(int i = -b ; i < b+1; i++)
    {
      for(int j = -b ; j < b+1; j++)
      {
        int curRow = row +i;
        int curCol = col +j;
        if(curRow> -1&& curRow< height && curCol> -1&& curCol< width)
          s+=imgIn[curRow*width + curCol];
      } 
      
    }
    s = s/((2*b+1)*(2*b+1));
    ImgOut[idx] = s; 
  }
}
__global__ void colorToGreykernel(uint8_t *ImgOut, const uint8_t* imgIn, int width, int height, int b)
{
  
  int col=blockIdx.x*blockDim.x+threadIdx.x;
  int row=blockIdx.y*blockDim.y+threadIdx.y;

  int idx = row*width + col;

  int s;
  if( (row<width) && (col <height))
  {
    ImgOut[idx] = imgIn[idx]; 
    /*
    s =0;

        for(int i = -b ; i < b+1; i++)
          for(int j = -b ; j < b+1; j++)
          {
            int curRow = row +i;
            int curCol = col +j;
            if(curRow> -1&& curRow< height && curCol> -1&& curCol< width)
              s+=imgIn[curRow*width + curCol];
          } 
          
        s = s/((2*b+1)*(2*b+1));
        ImgOut[idx] = s; 
    }
  */
  }
  
}


void blur_pgm(Image<int> src, Image<int> dst, int b)
{
  int *data;
  data = dst.getData();
  for(int i =b; i < dst.height();i++)
    for(int j = b ; j< dst.width();j++)
    {
      int s=0;

      for(int k = i-b; k<i+b ; k++)
        for(int l = j-b ; l<j+b ; l++)
          s+= src.get(k,l);
      s= s/((2*b+1)*(2*b+1));
      data[i*dst.width()+j]=s;
    }
    dst.write("image-grey2.pgm");

}



uint8_t * loadPPM(const char * file)
{
    uint8_t *data; 
    std::string control;
    int _col;
    int _row;
    std::ifstream is(file);
    is>>control;
    is>>_col>>_row;
    size_t channels = 3;

    data = (uint8_t*)malloc(_col*_row*channels);

    for (size_t i=0;i<_row;i++)
    for(size_t j=0;j<_col;j++)
    for(size_t c=0; c< channels;c++)
        is>>data[i*(_col*c)+j];


    std::ofstream os("output2.ppm");
    for (size_t i=0;i<_row;i++)
    for(size_t j=0;j<_col;j++)
        is>>data[i*_col+j];

    os<<control<<std::endl;
    os<<_col<<" "<<_row<<std::endl;




}
int main(int argc, char *argv[]) {
  srand( time( NULL ) );
    
  uint8_t *d_A, *d_B;
  uint8_t *h_A, *h_B;

  int * data;
  int *data2;
  //Image<int> src = Image<int>("P2", 512,512 );
  //Image<int> dst = Image<int>("P2", 512,512 );
  int n  = 512;
  int m  = 512;

  Image<int> src = Image<int>("P3", 512,512 );
  Image<int> dst = Image<int>("P3", 512,512 );

  int threads=16;
  //int size = m*n * sizeof(uint8_t);

  int size = 3*m*n *sizeof(uint8_t);

  dim3 DimGrid((n-1)/threads+1, (m-1)/threads+1,1);
  dim3 DimBlock(threads, threads,1);

  hipMalloc((void **) &d_A, size);    
  hipMalloc((void **) &d_B, size);

  h_A = (uint8_t*)malloc(size);
  h_B = (uint8_t*)malloc(size);

  //src.read("image-grey.pgm");
  src.read("image-color.ppm");

  data = src.getData();

  data2 = dst.getData();

  for(size_t i = 0 ; i < n*m; i++)
  {
    h_A[i]=(uint8_t )data[i];
  }
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);


  colorToGreykernel<<<DimGrid,DimBlock>>>(d_B,d_A, n,m,25);
  hipDeviceSynchronize();

  hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
  for(size_t i = 0 ; i < n*m; i++)
  {
    data2[i]=(int)h_B[i];
  }
  for(int i =0 ; i < n*m;i++)
      printf("%d ",data2[i]);
/*
  src.read("image-grey.pgm");
  data = src.getData(); 
  for(int i =0 ; i < n*m;i++)
    //printf("%d",data[i]);
    data[i]*=2;
  blur_pgm(src,  dst, 25);
  
  printf("ok");
  */
  dst.write("image-color2.ppm");

  loadPPM("image-color.ppm");

  hipFree(d_A); hipFree(d_B);
  free(h_A);  free(h_B); 

  cudaCheckError();
  return 0;
}
